#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mat.cuh"
#include <hip/hip_runtime.h>

#define ROW1 15
#define COL1 3 

#define ROW2 3
#define COL2 10 



int main() {
  // mat of size (ROW, COL)
  int * h_a = (int*) malloc (sizeof(int) * ROW1 * COL1);
  fill_rand_int(h_a, ROW1, COL1); 

  int * h_b = (int*) malloc (sizeof(int) * ROW2 * COL2);
  fill_rand_int(h_b, ROW2, COL2);

  int * p_h_c = (int*) malloc (sizeof(int) * ROW1 * COL2);

  int * d_a;
  int * d_b;
  int * d_c;

  //  __global__ 
  hipMalloc(&d_a, sizeof(int) * ROW1 * COL1);
  hipMalloc(&d_b, sizeof(int) * ROW2 * COL2);
  hipMalloc(&d_c, sizeof(int) * ROW1 * COL2);


  hipMemcpy(d_a, h_a, sizeof(int) * ROW1 * COL1, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sizeof(int) * ROW2 * COL2, hipMemcpyHostToDevice);


  dim3 gridDim (ROW1/B,COL2/B,1);
  dim3 blockDim (B,B,1);


  kernalMatMul<<<gridDim,blockDim>>>(d_a, d_b, d_c, ROW1, COL1, ROW2, COL2);

  hipDeviceSynchronize();

  hipMemcpy(p_h_c, d_c, sizeof(int) * ROW1 * COL2, hipMemcpyDeviceToHost);

  printf("parallel computation finished\n");

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  // printf("Printing result : \n");
  // printMat(p_h_c, ROW1, COL2);

  free(h_a);
  free(h_b);
  free(p_h_c);

  return 0;
}
