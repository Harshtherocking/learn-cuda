#include "hip/hip_runtime.h"
#include <stdio.h>
#include "./utils/mat.cuh"
#include "./utils/env.cuh"
#include <hip/hip_runtime.h>

#define N 4 

int main() {
  int * h_a = (int*) malloc (sizeof(int) * N * N);
  fill_rand_int(h_a, N, N); 

  printf("Printing A : \n");
  printMat(h_a, N, N);


  int * h_b = (int*) malloc (sizeof(int) * N * N);
  fill_rand_int(h_b, N, N);
  
  printf("Printing B: \n");
  printMat(h_b, N, N);

  int * h_c = (int*) malloc (sizeof(int) * N * N);

  int * d_a;
  int * d_b;
  int * d_c;

  //  __global__ 
  hipMalloc(&d_a, sizeof(int) * N * N);
  hipMalloc(&d_b, sizeof(int) * N * N);
  hipMalloc(&d_c, sizeof(int) * N * N);


  hipMemcpy(d_a, h_a, sizeof(int) * N * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sizeof(int) * N * N, hipMemcpyHostToDevice);


  dim3 gridDim (N/B, N/B,N/B);
  dim3 blockDim (B,B,1);


  Tiled_Mat_Multi<<<gridDim,blockDim>>>(d_a, d_b, d_c, N);

  hipDeviceSynchronize();

  hipMemcpy(h_c, d_c, sizeof(int) * N * N, hipMemcpyDeviceToHost);

  printf("parallel computation finished\n");

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  printf("Printing result : \n");
  printMat(h_c, N, N);

  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}
